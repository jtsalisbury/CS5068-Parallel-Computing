#include "hip/hip_runtime.h"
/* 
N-body simulation of a galaxy 
Authors: JT Salisbury, Sydney O'Connor, Kyle Bush, Caroline Northrop 
Parallel Computing 6068
*/

#include <cmath>
#include <math.h>
#include <vector>
#include <string>
#include "../../../cuda_by_example/common/book.h"
#include "../../../cuda_by_example/common/cpu_anim.h"

#define CONST_GRAVITY 0.00000000006673
#define CONST_TIME 1
#define CONST_MAX_NUM_POINTS 50
#define DIM 1024
#define TIME_OFFSET 10

// define structure containing all attributes of a simulation point
struct point {
	int id;
	float x_pos;
	float x_vel;
    float y_pos;
    float y_vel;
    float mass;
};

struct DataBlock {
    unsigned char *dev_bitmap;
    CPUAnimBitmap *bitmap;

    point *dev_sim_points_in;
    point *dev_sim_points_out;
    float *dev_total_force;
    float *dev_total_force_reduced;

    point sim_points_in[CONST_MAX_NUM_POINTS];
    point sim_points_out[CONST_MAX_NUM_POINTS];
    float total_force[CONST_MAX_NUM_POINTS * CONST_MAX_NUM_POINTS];
    float total_force_reduced[CONST_MAX_NUM_POINTS];
};

//TODO
// parse csv file 
/* assumptions:
variable names: id, x_pos, x_vel, y_pos, y_vel, mass
output: vector (in order above) of the elements 
*/

// reference: http://www.cplusplus.com/forum/beginner/193916/
void print_points(point p){
    std::cout << p.id << " "<< p.x_pos << " " << p.x_vel << " " << p.y_pos << " " << p.y_vel << " " << p.mass << "\n";
}

void parse_input() {
    //read file
    std::ifstream data("particles.csv");
    if (!data.is_open())
    {
        exit(EXIT_FAILURE);
    }
    std::string str;
    // getline(data, str); // skip the first line

    std::vector<point> my_points;
    int id;
    float x_pos;
    float x_vel;
    float y_pos;
    float y_vel;
    float mass;
    char delimiter;

    data.ignore(1000, '\n'); //ignore first line
    while(data >> id >> delimiter >> x_pos >> delimiter >> x_vel >> delimiter >> y_pos >> delimiter >> y_vel >> delimiter >> mass){
        point p;
        p.id;
        p.x_pos;
        p.x_vel;
        p.y_pos;
        p.y_vel;
        p.mass;

        my_points.push_back(p); //this line is causing issues
    }

    std::cout << "ID" << " X_POS" << " \n";
    for(int x(0); x<my_points.size(); ++x){
        print_points(my_points.at(x)); 
    }
}

// physics helper functions
__device__ float compute_force(float m1, float m2, float dist) {
	return CONST_GRAVITY * (m1 * m2/(dist * dist));
}

__device__ float compute_acceleration(float mass, float force) {
	return force/mass;
}

__device__ float compute_distance(float pos1, float pos2) {
	return abs(pos1 - pos2);
}

__device__ float compute_updated_pos(float pos, float vel, float acceleration) {
	return pos + (vel*CONST_TIME) +(.5 * acceleration * CONST_TIME * CONST_TIME);
}

__device__ float compute_updated_velocity(float vel, float acceleration) {
    return vel + (acceleration*CONST_TIME); 
}

// TODO: only performs x-component work so far, need to add y-component
__global__ void calculate_all_forces(point * sim_points_in, float * total_force) {
	// get the ids for each block and thread
	int k = blockIdx.x;
	int i = threadIdx.x;

    if (k == i) {
        // there is no force exerted on an object by the object itself
        total_force[k * CONST_MAX_NUM_POINTS + i] = 0;
    }
    else {
        // read the position and mass of the object
        float x_pos1 = sim_points_in[k].x_pos;
        float m1 = sim_points_in[k].mass;

        // obtain the positions of the 2nd object
        float x_pos2  = sim_points_in[i].x_pos;

        // calculate the distance between the 2 objects
        float dist = compute_distance(x_pos1, x_pos2);

        // obtain the masses of the 2 objects 
        float m2 = sim_points_in[i].mass;
                
        // calculate the force between the 2 objects
        float force_to_add = compute_force(m1, m2, dist);
                
        // add the force to the total force matrix
        total_force[k * CONST_MAX_NUM_POINTS + i] = force_to_add;
    }
	
}

__global__ void update_sim_points(float * total_force_reduced, point * sim_points_in, point * sim_points_out, unsigned char * bitmap) {
    // get the ids for each block and thread
    int k = blockIdx.x;

    // get initial position, velocity, and mass
    float x_pos1 = sim_points_in[k].x_pos;
    float x_vel1 = sim_points_in[k].x_vel;
    float m1 = sim_points_in[k].mass;

    // placeholders
    float y_pos1 = 2.0f;//sim_points_in[k].y_pos;
    
    // update the acceleration
    float acceleration = compute_acceleration(m1, total_force_reduced[k]);

    // update the velocity
    float updated_vel = compute_updated_velocity(x_vel1, acceleration);

    // update the position
    float updated_pos_x = compute_updated_pos(x_pos1, x_vel1, acceleration);
    
    // placeholder
    float updated_pos_y = 2.0f;//compute_updated_pos(y_pos1, y_vel1, acceleration);

    // store updated position and velocity
    sim_points_out[k].mass = m1;
    sim_points_out[k].x_vel = updated_vel;
    sim_points_out[k].x_pos = updated_pos_x;
        
    // update the bitmap only if in range
    if (x_pos1 < DIM && y_pos1 < DIM) {
        int oldOffset = x_pos1 + y_pos1 * gridDim.x;
        bitmap[oldOffset*4 + 0] = 0;
        bitmap[oldOffset*4 + 1] = 0;
        bitmap[oldOffset*4 + 2] = 0;
        bitmap[oldOffset*4 + 3] = 0;
    }

    if (x_pos2 < DIM && y_pos2 < DIM) {
        int newOffset = updated_pos_x + updated_pos_y * gridDim.x;
        bitmap[newOffset*4 + 0] = 255;
        bitmap[newOffset*4 + 1] = 255;
        bitmap[newOffset*4 + 2] = 255;
        bitmap[newOffset*4 + 3] = 255;
    }
}

// animation stuff
void generate_frame(DataBlock *d, int ticks) {
    // Only perform updates every N ticks
    if (ticks % TIME_OFFSET != 0) {
        return;
    }

    // allocate memory on GPU

    // copy simulation point array to GPU
    HANDLE_ERROR( hipMemcpy( d->dev_sim_points_in, d->sim_points_in, CONST_MAX_NUM_POINTS * sizeof(point),
    hipMemcpyHostToDevice ) );

    // run kernel - calculate all forces on every body in the simulation
    calculate_all_forces<<<CONST_MAX_NUM_POINTS, CONST_MAX_NUM_POINTS>>>(d->dev_sim_points_in, d->dev_total_force);

    // copy the total force matrix to CPU
    HANDLE_ERROR( hipMemcpy( d->total_force, d->dev_total_force, CONST_MAX_NUM_POINTS * CONST_MAX_NUM_POINTS * sizeof(float),
    hipMemcpyDeviceToHost ) );

    // perform a reduction
    for (int k = 0; k < CONST_MAX_NUM_POINTS; k++) {
        // reset the running sum to 0
        float running_sum = 0;
        for (int i = 0; i < CONST_MAX_NUM_POINTS; i++) {
            // add together all forces from every object
            running_sum += (d->total_force)[k * CONST_MAX_NUM_POINTS + i];
        } 
        // store the resulting total force in a new array
        (d->total_force_reduced)[k] = running_sum;
    }

    // copy the total force array to the GPU
    HANDLE_ERROR( hipMemcpy( d->dev_total_force_reduced, d->total_force_reduced, CONST_MAX_NUM_POINTS * sizeof(float),
    hipMemcpyHostToDevice ) );

    // run kernel - calculate updated position and velocity for the object
    update_sim_points<<<CONST_MAX_NUM_POINTS, 1>>>(d->dev_total_force_reduced, d->dev_sim_points_in, d->dev_sim_points_out, d->dev_bitmap);

    // copy simulation point array to CPU
    HANDLE_ERROR( hipMemcpy( d->sim_points_out, d->dev_sim_points_out, CONST_MAX_NUM_POINTS * sizeof(point),
    hipMemcpyDeviceToHost ) );

    HANDLE_ERROR( hipMemcpy( d->bitmap->get_ptr(), d->dev_bitmap, d->bitmap->image_size(), hipMemcpyDeviceToHost ) );

    // copy the output data to the input data
    memcpy(&(d->sim_points_in), &(d->sim_points_out), CONST_MAX_NUM_POINTS * sizeof(point));
}

void cleanup(DataBlock *d) {
    // free the memory allocated on the GPU
    HANDLE_ERROR( hipFree( d->dev_sim_points_in ) );
    HANDLE_ERROR( hipFree( d->dev_sim_points_out ) );
    HANDLE_ERROR( hipFree( d->dev_total_force ) );
    HANDLE_ERROR( hipFree( d->dev_total_force_reduced ) );

    HANDLE_ERROR( hipFree( d->dev_bitmap ) ); 
}

// TODO
// main function to perform physic operations 
int main() {
    //Note: when parsing input, put it in the data.sim_points_in array

    DataBlock data;

    CPUAnimBitmap bitmap(DIM, DIM, &data);
    data.bitmap = &bitmap;

    HANDLE_ERROR( hipMalloc( (void**)&(data.dev_bitmap), bitmap.image_size() ) );

    HANDLE_ERROR( hipMalloc( (void**)&(data.dev_sim_points_in), CONST_MAX_NUM_POINTS * sizeof(point) ) );
    HANDLE_ERROR( hipMalloc( (void**)&(data.dev_sim_points_out), CONST_MAX_NUM_POINTS * sizeof(point) ) );
    HANDLE_ERROR( hipMalloc( (void**)&(data.dev_total_force), CONST_MAX_NUM_POINTS * CONST_MAX_NUM_POINTS * sizeof(float) ) );
    HANDLE_ERROR( hipMalloc( (void**)&(data.dev_total_force_reduced), CONST_MAX_NUM_POINTS * sizeof(float) ) );

    bitmap.anim_and_exit( (void (*)(void*,int))generate_frame, (void (*)(void*))cleanup );

    return 0;
} 